
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int* devVal, int addEnd){
    *devVal= *devVal + addEnd;
    printf("Value after kernel is %d\n", *devVal);
}

int main(void){
    int* p0;
    int* p1;
    int* h1;
    int currentDev;
    size_t size = sizeof(int);

    hipSetDevice(0);

    hipMallocManaged(&p0, size);
    *p0 = 2;

    hipGetDevice(&currentDev);
    printf("Calling add on device: %d\n", currentDev);
    add<<<1,1>>>(p0, 2);

    hipDeviceSynchronize();

    hipSetDevice(1);

    hipMallocManaged(&p1, size);

    hipError_t memErr = hipMemcpyPeer(p1, 1, p0, 0, size);

    hipGetDevice(&currentDev);
    printf("Calling add on device: %d\n", currentDev);
    add<<<1,1>>>(p1, 3);

    hipDeviceSynchronize();

    printf("The final value is %d\n", *p1);
}