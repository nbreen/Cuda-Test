#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#define SRC_DEV 0
#define DST_DEV 1

#define DSIZE (8*1048576)

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


int main(int argc, char *argv[]){

  int disablePeer = 0;

  if (argc > 1) {
      disablePeer = 1;
  }
  int devcount;
  
  hipGetDeviceCount(&devcount);
  cudaCheckErrors("cuda failure");
  
  int srcdev = SRC_DEV;
  int dstdev = DST_DEV;
  
  if (devcount <= max(srcdev,dstdev)) {
      printf("not enough cuda devices for the requested operation\n"); 
      return 1;
  }

  int *d_s, *d_d, *h;
  
  int dsize = DSIZE*sizeof(int);
  
  h = (int *)malloc(dsize);
  
  if (h == NULL) {
    printf("malloc fail\n"); 
    return 1;
  }
  
  for (int i = 0; i < DSIZE; i++) {
      h[i] = i;
  }
  
  int canAccessPeer = 0;
  
  if (!disablePeer) {
    hipDeviceCanAccessPeer(&canAccessPeer, srcdev, dstdev);
  }

  hipSetDevice(srcdev);
  hipMalloc(&d_s, dsize);
  hipMemcpy(d_s, h, dsize, hipMemcpyHostToDevice);
  
  if (canAccessPeer) { 
    hipDeviceEnablePeerAccess(dstdev,0);
  }

  hipSetDevice(dstdev);
  hipMalloc(&d_d, dsize);
  hipMemset(d_d, 0, dsize);
  
  if (canAccessPeer) {
      hipDeviceEnablePeerAccess(srcdev,0);
  }

  cudaCheckErrors("hipMalloc/hipMemset fail");
  
  if (canAccessPeer) {
    printf("Timing P2P transfer");
  } else {
    printf("Timing ordinary transfer");
  }

  printf(" of %d bytes\n", dsize);
  
  hipEvent_t start, stop;
  hipEventCreate(&start); hipEventCreate(&stop);
  hipEventRecord(start);
  hipMemcpyPeer(d_d, dstdev, d_s, srcdev, dsize);
  cudaCheckErrors("hipMemcpyPeer fail");
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float et;
  hipEventElapsedTime(&et, start, stop);
  hipSetDevice(dstdev);
  hipMemcpy(h, d_d, dsize, hipMemcpyDeviceToHost);
  cudaCheckErrors("hipMemcpy fail");
  
  for (int i = 0; i < DSIZE; i++) { 
      if (h[i] != i) {
          printf("transfer failure\n"); 
          return 1;
      }
  }

  printf("transfer took %fms\n", et);
  
  return 0;
}